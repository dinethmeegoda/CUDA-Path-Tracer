#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__constant__ float cie_1964_dev_data[471][3];

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

 //Function to help with the conversion from wavelength to RGB
__host__ __device__ glm::vec3 wl_rgb(int wavelength) {
	wavelength -= 360;
    glm::vec3 xyz = (wavelength < 0 || wavelength > 470) ? glm::vec3(0.f) : glm::vec3(cie_1964_dev_data[wavelength][0], cie_1964_dev_data[wavelength][1], cie_1964_dev_data[wavelength][2]);
    float x = xyz.x;
    float y = xyz.y;
    float z = xyz.z;

    glm::vec3 rgb;
    rgb.r = 3.2404542 * x - 1.5371385 * y - 0.4985314 * z;
    rgb.g = -0.9692660 * x + 1.8760108 * y + 0.0415560 * z;
    rgb.b = 0.0556434 * x - 0.2040259 * y + 1.0572252 * z;
    return glm::clamp(rgb, 0.f, 1.f);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static Triangle* dev_triangles = NULL;
static Texture* dev_textures = NULL;
static glm::vec3* dev_texture_data = NULL;
static BVHNode* dev_bvhNodes = NULL;


void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

	// Constant Memory for CIE 1964 data for Wavelength Dispersion
	hipMemcpyToSymbol(HIP_SYMBOL(cie_1964_dev_data), cie_1964_host_data, 471 * sizeof(glm::vec3));
    checkCUDAError("pathtraceInit");

    // For Meshes

    if (scene->triangles.size() > 0) {

        hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
        hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

        hipMalloc(&dev_textures, scene->textures.size() * sizeof(Texture));
        hipMemcpy(dev_textures, scene->textures.data(), scene->textures.size() * sizeof(Texture), hipMemcpyHostToDevice);

        hipMalloc(&dev_texture_data, scene->textureData.size() * sizeof(glm::vec3));
        hipMemcpy(dev_texture_data, scene->textureData.data(), scene->textureData.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

        int num_nodes = scene->bvhNodes.size();
        hipMalloc(&dev_bvhNodes, num_nodes * sizeof(BVHNode));
        hipMemcpy(dev_bvhNodes, scene->bvhNodes.data(), num_nodes * sizeof(BVHNode), hipMemcpyHostToDevice);

    }

	checkCUDAError("pathtraceInit");

}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
	hipFree(dev_triangles);
	hipFree(dev_textures);
	hipFree(dev_texture_data);
    hipFree(dev_bvhNodes);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;

        // antialiasing by jittering the ray

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> u1_5(-1.5, 1.5);

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u1_5(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u1_5(rng) - (float)cam.resolution.y * 0.5f)
        );

        // wavelength setting
        thrust::uniform_real_distribution<float> u01(0, 1);
        segment.waveLength = u01(rng) * 470 + 360;
		segment.color = 3.0f * wl_rgb(segment.waveLength);

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    Triangle* tris,
    BVHNode* bvhnodes,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        glm::vec3 intersect_point;
        glm::vec3 normal;
		glm::vec2 uv;
        int meshId = 0;
  
#define BVH
#ifndef BVH
        float t;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        int temp_meshId = 0;
        // naive parse through global geoms
        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?
            else if (geom.type == MESH)
            {
				t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tris, geom.triangleStart, geom.triangleEnd);
            }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
				uv = tmp_uv;
                meshId = temp_meshId;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
			pathSegment.remainingBounces = 0;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
			intersections[path_index].uv = uv;
            intersections[path_index].hasUV = geoms[hit_geom_index].usesUVs;
            if (geoms[hit_geom_index].usesUVs) {
                intersections[path_index].texid = geoms[hit_geom_index].textureStart;
            }
        }
#endif
#ifdef BVH
        intersections[path_index].t = bvhMeshIntersectionTest(geoms, pathSegment.ray, intersect_point, normal, uv, tris, bvhnodes, meshId);
		if (intersections[path_index].t < 0.0f) {
            // Hit nothing
			intersections[path_index].t = -1.0f;
			pathSegment.remainingBounces = 0;
		}
        intersections[path_index].materialId = geoms[meshId].materialid;
        intersections[path_index].surfaceNormal = normal;
        intersections[path_index].uv = uv;
        intersections[path_index].hasUV = geoms[meshId].usesUVs;
        if (geoms[meshId].usesUVs) {
            intersections[path_index].texid = geoms[meshId].textureStart;
        }

#endif
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                //pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    Texture* textureMaps,
    glm::vec3* textureColors)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
		// Check if the ray is terminated
		if (pathSegments[idx].remainingBounces <= 0)
        {
			return;
		}

        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
            // Set up the RNG
            Material material = materials[intersection.materialId];

            // If the material indicates that the object was a light, "light" the ray and terminate the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (material.color * material.emittance);
				pathSegments[idx].remainingBounces = 0;
            }
			// Otherwise, bounce the ray
            else {

                thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
                thrust::uniform_real_distribution<float> u01(0, 1);

                // Get the ray
				Ray& ray = pathSegments[idx].ray;

				// Get the intersection point
				glm::vec3 intersect = ray.origin + ray.direction * intersection.t;

				glm::vec3 textureCol = glm::vec3(-1.0f);

				// Get the texture color
                if (intersection.hasUV) {
					int x = glm::min(textureMaps[intersection.texid].width * intersection.uv.x, textureMaps[intersection.texid].width - 1.0f);
					int y = glm::min(textureMaps[intersection.texid].height * intersection.uv.y, textureMaps[intersection.texid].height - 1.0f);
					int idx = textureMaps[intersection.texid].width * y + x + textureMaps[intersection.texid].startIndex;
					textureCol = textureColors[idx];
                }

                scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng, textureCol);

            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

// Predicate for stream compaction
struct is_path_terminated
{
	__host__ __device__ bool operator()(const PathSegment& path)
	{
        return path.remainingBounces > 0;
	}
};

// Predicate for sorting by material
struct compare_material
{
	__host__ __device__ bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b)
	{
		return a.materialId < b.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
			dev_triangles,
            dev_bvhNodes,
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        /*shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> >(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );*/

#define MATERIAL_SORTING 1
#if MATERIAL_SORTING
		// Sort the paths by material
		thrust::device_ptr<ShadeableIntersection> dev_intersections_ptr(dev_intersections);
		thrust::device_ptr<PathSegment> dev_paths_ptr(dev_paths);
		thrust::stable_sort_by_key(dev_intersections_ptr, dev_intersections_ptr + num_paths, dev_paths_ptr, compare_material());
#endif

		shadeMaterial <<<numblocksPathSegmentTracing, blockSize1d >>> (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_textures,
			dev_texture_data
			);

#define STREAM_COMPACTION 1
#if STREAM_COMPACTION
		// Stream compaction using thrust
        thrust::device_ptr<PathSegment> dev_compaction_paths(dev_paths);
		
		thrust::device_ptr<PathSegment> new_paths_end = thrust::stable_partition(thrust::device, dev_compaction_paths, dev_compaction_paths + num_paths, is_path_terminated());
		num_paths = new_paths_end.get() - dev_paths;
#endif

		iterationComplete = (depth >= traceDepth || num_paths == 0);

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
